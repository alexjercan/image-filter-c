#include "hip/hip_runtime.h"
#include "util.h"

#define NUM_THREADS_PER_BLOCK 32

__global__ void applyKernel(unsigned char *d_img_bytes, int width, int height,
                            int channels, float *d_kernel, int size,
                            unsigned char *d_out_bytes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= width || y >= height || c >= channels)
        return;

    float accum = 0.0f;

    for (int ky = 0; ky < size; ky++) {
        for (int kx = 0; kx < size; kx++) {
            int img_x = x + kx - size / 2;
            int img_y = y + ky - size / 2;
            int k_x = size - kx - 1;
            int k_y = size - ky - 1;

            unsigned char pixel = 0;
            if (img_x >= 0 && img_x < width && img_y >= 0 && img_y < height &&
                c >= 0 && c < channels) {
                pixel = d_img_bytes[(img_y * width + img_x) * channels + c];
            }

            float value = 0.0f;
            if (k_x >= 0 && k_x < size && k_y >= 0 && k_y < size) {
                value = d_kernel[k_y * size + k_x];
            }

            accum += pixel * value;
        }
    }

    if (accum < 0.0f) {
        accum = 0.0f;
    } else if (accum > 255.0f) {
        accum = 255.0f;
    }
    d_out_bytes[(y * width + x) * channels + c] = (unsigned char)accum;
}

static int imageApplyKernel(unsigned char *d_img_bytes, int width, int height,
                            int channels, float *d_kernel, int size,
                            unsigned char *d_out_bytes) {
    dim3 threadsPerBlock(NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK, 1);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (channels + threadsPerBlock.z - 1) / threadsPerBlock.z);

    applyKernel<<<numBlocks, threadsPerBlock>>>(
        d_img_bytes, width, height, channels, d_kernel, size, d_out_bytes);
    hipDeviceSynchronize();

    return 0;
}

extern "C" {
#include "image.h"

int image_apply_kernel_cuda_wrapper(struct image *img, struct kernel *k,
                                    struct image *out, int repeats) {
    int result = 0;
    unsigned char *d_img_bytes = NULL;
    unsigned char *d_out_bytes = NULL;
    float *d_kernel = NULL;

    hipError_t error;

    error = hipMalloc((void **)&d_img_bytes, img->width * img->height *
                                                  img->channels *
                                                  sizeof(unsigned char));
    if (error != hipSuccess) {
        LOG_ERROR("hipMalloc failed: %s\n", hipGetErrorString(error));
        return_defer(1);
    }

    error = hipMalloc((void **)&d_out_bytes, img->width * img->height *
                                                  img->channels *
                                                  sizeof(unsigned char));
    if (error != hipSuccess) {
        LOG_ERROR("hipMalloc failed: %s\n", hipGetErrorString(error));
        return_defer(1);
    }

    error = hipMalloc((void **)&d_kernel, k->size * k->size * sizeof(float));
    if (error != hipSuccess) {
        LOG_ERROR("hipMalloc failed: %s\n", hipGetErrorString(error));
        return_defer(1);
    }

    error = hipMemcpy(d_img_bytes, img->bytes,
                       img->width * img->height * img->channels *
                           sizeof(unsigned char),
                       hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        LOG_ERROR("hipMalloc failed: %s\n", hipGetErrorString(error));
        return_defer(1);
    }

    error = hipMemcpy(d_kernel, k->values, k->size * k->size * sizeof(float),
                       hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        LOG_ERROR("hipMalloc failed: %s\n", hipGetErrorString(error));
        return_defer(1);
    }

    for (int i = 0; i < repeats; i++) {
        imageApplyKernel(d_img_bytes, img->width, img->height, img->channels,
                         d_kernel, k->size, d_out_bytes);
        hipMemcpy(d_img_bytes, d_out_bytes,
                   img->width * img->height * img->channels *
                       sizeof(unsigned char),
                   hipMemcpyDeviceToDevice);
    }

    error = hipMemcpy(out->bytes, d_out_bytes,
                       img->width * img->height * img->channels *
                           sizeof(unsigned char),
                       hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        LOG_ERROR("hipMalloc failed: %s\n", hipGetErrorString(error));
        return_defer(1);
    }

defer:
    if (d_img_bytes)
        hipFree(d_img_bytes);
    if (d_out_bytes)
        hipFree(d_out_bytes);

    return result;
}
}
